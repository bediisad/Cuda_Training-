#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void bdkernel1()
{
    printf("Hello World from GPU-1\n");
}

__global__ void bdkernel2()
{
    printf("Hello World from GPU-2\n");
}
int main()
{
    bdkernel1<<<1,1>>>();
    bdkernel2<<<1,1>>>();
    hipDeviceSynchronize();

    printf("CPU\n");
    return 0;
}