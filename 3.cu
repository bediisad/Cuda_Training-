#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void bdkernel()
{
    printf("Hello World from GPU\n");
}
int main()
{
    bdkernel<<<1,1>>>();
    bdkernel<<<1,1>>>();
    bdkernel<<<1,1>>>();

    printf("CPU\n");
    return 0;
}