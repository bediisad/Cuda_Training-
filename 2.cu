#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void bdkernel()
{
    printf("Hello World from GPU\n");
}
int main()
{
    bdkernel<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}